#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <opencv2/opencv.hpp>

#define BLOCK_SIZE 16
texture<unsigned char, 2, hipReadModeElementType> inTexture;

__global__ void gpuCalculation(unsigned char* output, int width, int height) {
	int txIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int tyIndex = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned char res[9];
	unsigned char tmp;
	if ((txIndex < width) && (tyIndex < height)){
		for (int i = 0; i < 3; i++)
			for (int j = 0; j < 3; j++)
				res[i * 3 + j] = tex2D(inTexture, txIndex + i-1, tyIndex + j-1);
		for (int i = 0; i < 8; i++) {
			for (int j = 0; j < 8 - i; j++) {
				if (res[j] > res[j + 1]) {
					tmp = res[j];
					res[j] = res[j + 1];
					res[j + 1] = tmp;
				}
			}
		}
		output[tyIndex*width + txIndex] = res[4];
	}
}

void pepper(const cv::Mat & input, cv::Mat & output) {
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int gray_size = input.step*input.rows;
	size_t pitch;
	unsigned char *d_input = NULL;
	unsigned char *d_output;
	hipMallocPitch(&d_input, &pitch, sizeof(unsigned char)*input.step, input.rows);
	hipMemcpy2D(d_input, pitch, input.ptr(), sizeof(unsigned char)*input.step, sizeof(unsigned char)*input.step, input.rows, hipMemcpyHostToDevice);
	hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
	hipBindTexture2D(0, inTexture, d_input, desc, input.step, input.rows, pitch);
	hipMalloc<unsigned char>(&d_output, gray_size);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((input.cols + BLOCK_SIZE - 1) / BLOCK_SIZE, (input.rows + BLOCK_SIZE - 1) / BLOCK_SIZE);
	hipEventRecord(start, 0);
	gpuCalculation <<<grid, block >>> (d_output, input.cols, input.rows);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipMemcpy(output.ptr(), d_output, gray_size, hipMemcpyDeviceToHost);
	hipUnbindTexture(inTexture);
	hipFree(d_input);
	hipFree(d_output);
	hipEventElapsedTime(&time, start, stop);
	std::cout << "Time for the GPU: " << time << " ms" << std::endl;
}