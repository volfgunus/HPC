#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <opencv2/opencv.hpp>

#define BLOCK_SIZE 16

texture<unsigned char, 2, hipReadModeElementType> inTexture;


__global__ void gpuCalculation(unsigned char* output, int width, int height) {
	int txIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int tyIndex = threadIdx.y + blockIdx.y * blockDim.y;
	if ((txIndex < width) && (tyIndex < height)) {
		float txnorm = txIndex / height +0.5f;
		float tynorm = tyIndex / width +0.5f;
		output[tyIndex*width + txIndex] = tex2D(inTexture, txnorm, tynorm);
	}
}

void resize(const cv::Mat & input, cv::Mat & output) {
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int gray_size = output.step*output.rows;
	size_t pitch;
	unsigned char *d_input = NULL;
	unsigned char *d_output;
	hipMallocPitch(&d_input, &pitch, sizeof(unsigned char)*input.step, input.rows);
	hipMemcpy2D(d_input, pitch, input.ptr(), sizeof(unsigned char)*input.step, sizeof(unsigned char)*input.step, input.rows, hipMemcpyHostToDevice);
	hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
	inTexture.addressMode[0] = hipAddressModeClamp;
	inTexture.addressMode[1] = hipAddressModeClamp;
	inTexture.filterMode = hipFilterModeLinear;
	inTexture.normalized = true;
	hipBindTexture2D(0, inTexture, d_input, desc, input.step, input.rows, pitch);
	hipMalloc<unsigned char>(&d_output, gray_size);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((output.cols + BLOCK_SIZE - 1) / BLOCK_SIZE, (output.rows + BLOCK_SIZE - 1) / BLOCK_SIZE);
	hipEventRecord(start, 0);
	gpuCalculation << <grid, block >> > (d_output, output.cols, output.rows);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipMemcpy(output.ptr(), d_output, gray_size, hipMemcpyDeviceToHost);
	hipUnbindTexture(inTexture);
	hipFree(d_input);
	hipFree(d_output);
	hipEventElapsedTime(&time, start, stop);
	std::cout << "Time for the GPU: " << time << " ms" << std::endl;
}